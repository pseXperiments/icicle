#include "hip/hip_runtime.h"


#define SHMEM_SIZE 64
#define MAX_SHMEM_LOG_SIZE 6
#define FIELD_ID BN254
#define CURVE_ID BN254

#include "../ntt/kernel_ntt.cu"
// static inline __device__ uint32_t bit_rev(uint32_t num, uint32_t log_size) { return __brev(num) >> (32 - log_size); }

// template <typename S>
// __global__ void inplace_rbo(S* arr, int size){
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	S temp = arr[tid];
// 	arr[tid] = arr[bit_rev(tid)];
// 	arr[bit_rev(tid)] = temp;
// }

template <typename S>
__global__ void mult_and_reduce_double(S *v, S *v_r, S alpha1, S alpha2, int stride, int jump_size) {
	// Allocate shared memory
	__shared__ S partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements AND do first add of reduction
	// Vector now 2x as long as number of threads, so scale i
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	// Store first partial result instead of just the elements
	// partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];
	// partial_sum[threadIdx.x] = (S::one() - alpha) * v[2*i] + alpha * v[2*i+1] + (S::one() - alpha) * v[2*(i + blockDim.x)] + alpha * v[2*(i + blockDim.x)+1];
	S e1 = v[i*stride];
	S e2 = v[(i+2*jump_size)*stride];
	S e3 = v[(i+jump_size)*stride];
	S e4 = v[(i+3*jump_size)*stride];
	S f1 = e1 + alpha1 * (e3 - e1) + alpha2 * (e2 - e1) + alpha1 * alpha2 * (e1 + e4 - e2 - e3);

	e1 = v[(i+blockDim.x)*stride];
	e2 = v[(i+blockDim.x+2*jump_size)*stride];
	e3 = v[(i+blockDim.x+jump_size)*stride];
	e4 = v[(i+blockDim.x+3*jump_size)*stride];
	S f2 = e1 + alpha1 * (e3 - e1) + alpha2 * (e2 - e1) + alpha1 * alpha2 * (e1 + e4 - e2 - e3);
	// S e1 = v[2*i] + (v[2*i+1] - v[2*i]);
	// S e2 = v[2*(i + blockDim.x)] + (v[2*(i + blockDim.x)+1] - v[2*(i + blockDim.x)]);
	// partial_sum[threadIdx.x] = v[2*i] + v[2*(i + blockDim.x)] + alpha * (v[2*i+1] - v[2*i] + v[2*(i + blockDim.x)+1] - v[2*(i + blockDim.x)]);
	partial_sum[threadIdx.x] = f1 + f2;
	// __syncthreads();
	v[i*stride] = f1;
	v[(i + blockDim.x)*stride] = f2;
	// for (int j = 0; j < 2; j++)
	// {
	// 	partial_sum[threadIdx.x] = partial_sum[threadIdx.x] * partial_sum[threadIdx.x];
	// }
	
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
	// for (int s = blockDim.x / 2; s > 1; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] = partial_sum[threadIdx.x] + partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	// if (threadIdx.x < nof_results) {
	if (threadIdx.x == 0) {
		// printf("debug tid %d, val %d\n", threadIdx.x, partial_sum[threadIdx.x]);
		// v_r[nof_results*blockIdx.x + threadIdx.x] = partial_sum[threadIdx.x];
		v_r[i*stride] = partial_sum[0];
	}
}

template <typename S>
__global__ void mult_and_reduce(S *v, S *v_r, S alpha, int stride, int jump_size) {
	// Allocate shared memory
	__shared__ S partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements AND do first add of reduction
	// Vector now 2x as long as number of threads, so scale i
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	// Store first partial result instead of just the elements
	// partial_sum[threadIdx.x] = v[i] + v[i + blockDim.x];
	// partial_sum[threadIdx.x] = (S::one() - alpha) * v[2*i] + alpha * v[2*i+1] + (S::one() - alpha) * v[2*(i + blockDim.x)] + alpha * v[2*(i + blockDim.x)+1];
	S e1 = v[i*stride] + alpha * (v[(i+jump_size)*stride] - v[i*stride]);
	S e2 = v[(i + blockDim.x)*stride] + alpha * (v[(i + blockDim.x+jump_size)*stride] - v[(i + blockDim.x)*stride]);
	// S e1 = v[2*i] + (v[2*i+1] - v[2*i]);
	// S e2 = v[2*(i + blockDim.x)] + (v[2*(i + blockDim.x)+1] - v[2*(i + blockDim.x)]);
	// partial_sum[threadIdx.x] = v[2*i] + v[2*(i + blockDim.x)] + alpha * (v[2*i+1] - v[2*i] + v[2*(i + blockDim.x)+1] - v[2*(i + blockDim.x)]);
	partial_sum[threadIdx.x] = e1 + e2;
	// __syncthreads();
	v[i*stride] = e1;
	v[(i + blockDim.x)*stride] = e2;
	// for (int j = 0; j < 2; j++)
	// {
	// 	partial_sum[threadIdx.x] = partial_sum[threadIdx.x] * partial_sum[threadIdx.x];
	// }
	
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
	// for (int s = blockDim.x / 2; s > 1; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] = partial_sum[threadIdx.x] + partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	// if (threadIdx.x < nof_results) {
	if (threadIdx.x == 0) {
		// printf("debug tid %d, val %d\n", threadIdx.x, partial_sum[threadIdx.x]);
		// v_r[nof_results*blockIdx.x + threadIdx.x] = partial_sum[threadIdx.x];
		v_r[i*stride] = partial_sum[0];
	}
}


template <typename S>
__global__ void sum_reduction(S *v, S *v_r, int stride) {
	// Allocate shared memory
	__shared__ S partial_sum[SHMEM_SIZE];

	// Calculate thread ID
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Load elements AND do first add of reduction
	// Vector now 2x as long as number of threads, so scale i
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	// Store first partial result instead of just the elements
	partial_sum[threadIdx.x] = v[i*stride] + v[(i + blockDim.x)*stride];
	__syncthreads();

	// Start at 1/2 block stride and divide by two each iteration
	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
	// for (int s = blockDim.x / 2; s > 1; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] = partial_sum[threadIdx.x] + partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	// Let the thread 0 for this block write it's result to main memory
	// Result is inexed by this block
	// if (threadIdx.x < nof_results) {
	if (threadIdx.x == 0) {
		// printf("debug tid %d, val %d\n", threadIdx.x, partial_sum[threadIdx.x]);
		// v_r[blockIdx.x] = partial_sum[0];
		v_r[i*stride] = partial_sum[0];
		// v_r[nof_results*blockIdx.x + threadIdx.x] = partial_sum[threadIdx.x];
	}
}

template <typename S>
__global__ void update_evals_kernel(S* evals, S alpha, int poly_size, int poly_shift, int nof_ploys){
  int threads_per_poly = poly_size/2;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= threads_per_poly*nof_ploys) return;
	int poly_id = tid / threads_per_poly;
	int eval_id = tid % threads_per_poly;
  // evals[tid] = (S::one() - alpha) * evals[2*tid] + alpha * evals[2*tid+1];
  // evals[tid] =  evals[2*tid] + (evals[2*tid+1] - evals[2*tid]);
	// if (tid==0) printf("%d, %d, %d, %d, %d\n", poly_size, poly_id, eval_id, poly_id*poly_size*2+eval_id, poly_id*poly_size*2+eval_id+poly_size);
	// if (tid==0) printf("what12 %d %d\n",evals[poly_id*poly_size*2 + eval_id], evals[poly_id*poly_size*2 + eval_id+poly_size]);
  evals[poly_id*poly_shift + eval_id] =  evals[poly_id*poly_shift+eval_id] + alpha * (evals[poly_id*poly_shift+eval_id+threads_per_poly] - evals[poly_id*poly_shift+eval_id]);
	// if (tid==0) printf("what %d\n",evals[poly_id*poly_size*2 + eval_id]);
  // evals[tid] = (1 - alpha) * evals[2*tid] + alpha * evals[2*tid+1];
}

template <typename S>
__global__ void update_evals_double_kernel(S* evals, S alpha1, S alpha2, int poly_size, int poly_shift, int nof_ploys){
  int threads_per_poly = poly_size/4;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= threads_per_poly*nof_ploys) return;
	int poly_id = tid / threads_per_poly;
	int eval_id = tid % threads_per_poly;
	S e1 = evals[poly_id*poly_shift+eval_id];
	S e2 = evals[poly_id*poly_shift+eval_id+2*threads_per_poly];
	S e3 = evals[poly_id*poly_shift+eval_id+threads_per_poly];
	S e4 = evals[poly_id*poly_shift+eval_id+3*threads_per_poly];
  evals[poly_id*poly_shift + eval_id] =  e1 + alpha1 * (e3 - e1) + alpha2 * (e2 - e1) + alpha1 * alpha2 * (e1 + e4 - e2 - e3);
}

template <typename S>
void accumulate(S* in, S* out, int log_size, int nof_results, int nof_rounds, hipStream_t stream){
  int nof_steps = (log_size - nof_rounds) / MAX_SHMEM_LOG_SIZE;
  int last_step_size = (log_size - nof_rounds) % MAX_SHMEM_LOG_SIZE;
	// printf("a nof steps %d last size %d\n", nof_steps, last_step_size);
  for (int i = 0; i < nof_steps; i++)
  {
    sum_reduction<<<(1<<(log_size - nof_rounds - (MAX_SHMEM_LOG_SIZE)*(i+1))) * nof_results * nof_rounds, SHMEM_SIZE/2,0,stream>>>(i? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*i));
		// printf("a nof blocks %d\n", 1<<(log_size -(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// hipDeviceSynchronize();
  	// printf("cuda err %d\n", hipGetLastError());
  }
  if (last_step_size) sum_reduction<<<nof_results * nof_rounds, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*nof_steps));
	// hipDeviceSynchronize();
  // printf("cuda err last %d\n", hipGetLastError());
}

template <typename S>
void mult_and_accumulate_double(S* in, S* out, int log_size, S alpha1, S alpha2, int nof_results, hipStream_t stream){
  int nof_steps = (log_size - 2) / MAX_SHMEM_LOG_SIZE;
  int last_step_size = (log_size - 2) % MAX_SHMEM_LOG_SIZE;
	// printf("m nof steps %d last size %d\n", nof_steps, last_step_size);
  for (int i = 0; i < nof_steps; i++)
  {
		if (i) sum_reduction<<<(1<<(log_size - 2 - (MAX_SHMEM_LOG_SIZE)*(i+1))) * nof_results * 2, SHMEM_SIZE/2,0,stream>>>(i? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*i));
    else mult_and_reduce_double<<<(1<<(log_size - 2 - (MAX_SHMEM_LOG_SIZE)*(i+1))) * nof_results * 2, SHMEM_SIZE/2,0,stream>>>(i? out : in, out, alpha1, alpha2, 1<<(MAX_SHMEM_LOG_SIZE*i), 1<<log_size);
		// if (i) printf("r nof blocks %d\n", 1<<(log_size-(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// else printf("m nof blocks %d\n", 1<<(log_size-(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// hipDeviceSynchronize();
  	// printf("cuda err %d\n", hipGetLastError());
  }
  if (last_step_size) {
		if (nof_steps) sum_reduction<<<nof_results * 2, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*nof_steps));
		else mult_and_reduce_double<<<nof_results * 2, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, alpha1, alpha2, 1<<(MAX_SHMEM_LOG_SIZE*nof_steps), 1<<(last_step_size+2));
		// if (nof_steps) printf("r last");
		// else printf("m last");
	} 
	hipDeviceSynchronize();
	// printf("nof res %d last_step_size %d\n", nof_results, last_step_size);
  // printf("cuda err last %d\n", hipGetLastError());
}

template <typename S>
void mult_and_accumulate(S* in, S* out, int log_size, S alpha, int nof_results, hipStream_t stream){
  int nof_steps = (log_size - 1) / MAX_SHMEM_LOG_SIZE;
  int last_step_size = (log_size - 1) % MAX_SHMEM_LOG_SIZE;
	// printf("m nof steps %d last size %d\n", nof_steps, last_step_size);
  for (int i = 0; i < nof_steps; i++)
  {
		if (i) sum_reduction<<<(1<<(log_size - 1 - (MAX_SHMEM_LOG_SIZE)*(i+1))) * nof_results, SHMEM_SIZE/2,0,stream>>>(i? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*i));
    else mult_and_reduce<<<(1<<(log_size - 1 - (MAX_SHMEM_LOG_SIZE)*(i+1))) * nof_results, SHMEM_SIZE/2,0,stream>>>(i? out : in, out, alpha, 1<<(MAX_SHMEM_LOG_SIZE*i), 1<<log_size);
		// if (i) printf("r nof blocks %d\n", 1<<(log_size-(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// else printf("m nof blocks %d\n", 1<<(log_size-(MAX_SHMEM_LOG_SIZE)*(i+1)));
		// hipDeviceSynchronize();
  	// printf("cuda err %d\n", hipGetLastError());
  }
  if (last_step_size) {
		if (nof_steps) sum_reduction<<<nof_results, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, 1<<(MAX_SHMEM_LOG_SIZE*nof_steps));
		else mult_and_reduce<<<nof_results, 1<<(last_step_size-1), 0,stream>>>(nof_steps? out : in, out, alpha, 1<<(MAX_SHMEM_LOG_SIZE*nof_steps), 1<<(last_step_size+1));
		// if (nof_steps) printf("r last");
		// else printf("m last");
	} 
	// hipDeviceSynchronize();
	// printf("nof res %d last_step_size %d\n", nof_results, last_step_size);
  // printf("cuda err last %d\n", hipGetLastError());
}

template <typename S>
 __launch_bounds__(1)
__global__ void add_to_trace(S* trace, S* vals, int stride, int round_num, int nof_results){
	for (int i = 0; i < nof_results; i++)
	{
		trace[nof_results*round_num+1+i] = vals[i*stride];
	}
	// for (int i = 0; i < nof_results; i++)
	// {
	// 	trace[nof_results*round_num+1+i] = vals[i];
	// }
	  // trace[2*round_num+1] = vals[0];
    // trace[2*round_num+2] = vals[1];
		// printf("%d  %d\n", vals[0], vals[1]);
}

			// T[(nof_polys+1)*(nof_polys+1)*p+1] = T[(nof_polys+1)*(nof_polys+1)*p+1] + rp[0];
			// T[(nof_polys+1)*(nof_polys+1)*p+2] = T[(nof_polys+1)*(nof_polys+1)*p+2] + rp[1];
			// T[(nof_polys+1)*(nof_polys+1)*p+3] = T[(nof_polys+1)*(nof_polys+1)*p+3] + rp[2];
			// T[(nof_polys+1)*(nof_polys+1)*p+4] = T[(nof_polys+1)*(nof_polys+1)*p+4] + rp[3];
			// if (nof_polys > 1) {
			// 	T[(nof_polys+1)*(nof_polys+1)*p+5] = T[(nof_polys+1)*(nof_polys+1)*p+5] + rp[4];
			// 	T[(nof_polys+1)*(nof_polys+1)*p+6] = T[(nof_polys+1)*(nof_polys+1)*p+6] + rp[5];
			// 	T[(nof_polys+1)*(nof_polys+1)*p+7] = T[(nof_polys+1)*(nof_polys+1)*p+7] + rp[6];
			// 	T[(nof_polys+1)*(nof_polys+1)*p+8] = T[(nof_polys+1)*(nof_polys+1)*p+8] + rp[7];
			// 	T[(nof_polys+1)*(nof_polys+1)*p+9] = T[(nof_polys+1)*(nof_polys+1)*p+9] + rp[8];
			// }

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void combinations_kernel3(S* in, S* out, int poly_size, int poly_shift){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= poly_size/2) return;
	S rp[4] = {S::one(), S::one(), S::one(), S::one()};
	S e1, e2;
	#pragma unroll
	for (int l = 0; l < 3; l++)
	{
	  e1 = in[l*poly_shift + tid];
	  e2 = in[l*poly_shift + tid + poly_size/2];
		rp[0] = l? rp[0]*e1 : e1; //k=0
		rp[1] = l? rp[1]*e2 : e2; //k=1
		rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
		// rp[3] = l? rp[3]*(e1 + e1 - e2) : (e1 + e1 - e2); //k=-1
		rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
	}
	out[tid] = rp[0];
	out[tid + 1*poly_size/2] = rp[1];
	out[tid + 2*poly_size/2] = rp[2];
	out[tid + 3*poly_size/2] = rp[3];
}

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void combinations_kernel(S* in, S* out, int poly_size, int poly_shift, int nof_polys){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= poly_size/2) return;
	S rp[5] = {S::one(), S::one(), S::one(), S::one(), S::one()}; //TODO: generalize - make template version
	S e1, e2;
	#pragma unroll
	for (int l = 0; l < nof_polys; l++)
	{
	  e1 = in[l*poly_shift + tid];
	  e2 = in[l*poly_shift + tid + poly_size/2];
		rp[0] = l? rp[0]*e1 : e1; //k=0
		rp[1] = l? rp[1]*e2 : e2; //k=1
		if (nof_polys == 1) continue;
		rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
		if (nof_polys == 2) continue;
		// rp[3] = l? rp[3]*(e1 + e1 - e2) : (e1 + e1 - e2); //k=-1
		rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
		if (nof_polys == 3) continue;
		rp[4] = l? rp[4]*(e2 + e2 + e2 + e2 - e1 - e1 - e1) : (e2 + e2 + e2 + e2 - e1 - e1 - e1); //k=4 TODO: save addition using extra reg?
	}
	out[tid] = rp[0];
	out[tid + 1*poly_size/2] = rp[1];
	if (nof_polys == 1) return;
	out[tid + 2*poly_size/2] = rp[2];
	if (nof_polys == 2) return;
	out[tid + 3*poly_size/2] = rp[3];
	if (nof_polys == 3) return;
	out[tid + 4*poly_size/2] = rp[4];
}

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void combinations_double_kernel(S* in, S* out, int poly_size, int poly_shift, int nof_polys){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= poly_size/4) return;
	S rp[9] = {S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one()}; //TODO: generalize - make template version
	S e1, e2, e3, e4;
	#pragma unroll 1
	for (int l = 0; l < nof_polys; l++)
	{
	  e1 = in[l*poly_shift + tid];
	  e2 = in[l*poly_shift + tid + poly_size/2];
	  e3 = in[l*poly_shift + tid + poly_size/4];
	  e4 = in[l*poly_shift + tid + 3*poly_size/4];
		rp[0] = l? rp[0]*e1 : e1; //k=0,0
		rp[1] = l? rp[1]*e2 : e2; //k=0,1
		rp[2] = l? rp[2]*e3 : e3; //k=1,0
		rp[3] = l? rp[3]*e4 : e4; //k=1,1
		if (nof_polys == 1) continue;
		rp[4] = l? rp[4]*(e2+e2-e1) : (e2+e2-e1); //k=0,2
		rp[5] = l? rp[5]*(e3+e3-e1) : (e3+e3-e1); //k=2,0
		rp[6] = l? rp[6]*(e4+e4-e3) : (e4+e4-e3); //k=1,2
		rp[7] = l? rp[7]*(e4+e4-e2) : (e4+e4-e2); //k=2,1
		rp[8] = l? rp[8]*(e4+e4+e4+e4+e1-e2-e2-e3-e3) : (e4+e4+e4+e4+e1-e2-e2-e3-e3); //k=2,2
	}
	out[tid] = rp[0];
	out[tid + 1*poly_size/4] = rp[2];
	out[tid + 2*poly_size/4] = rp[1];
	out[tid + 3*poly_size/4] = rp[3];
	if (nof_polys == 1) return;
	out[tid + 4*poly_size/4] = rp[4];
	out[tid + 5*poly_size/4] = rp[5];
	out[tid + 6*poly_size/4] = rp[6];
	out[tid + 7*poly_size/4] = rp[7];
	out[tid + 8*poly_size/4] = rp[8];
}


template <typename S>
__global__ void combinations_double_test(S* in, S* out){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	S rp[4] = {S::one(), S::one(), S::one(), S::one()};
	rp[0] = in[tid];
	rp[1] = in[tid+1];
	rp[2] = in[tid+2];
	rp[3] = in[tid+3];
	out[tid] = rp[0] * rp[1];
	out[tid+1] = rp[2] * rp[3];
	out[tid+2] = rp[2] * rp[1];
	out[tid+3] = rp[0] * rp[3];
	out[tid+4] = rp[0] * rp[2];
	out[tid+5] = rp[0] * rp[0];
	out[tid+6] = rp[1] * rp[1];
	out[tid+7] = rp[1] * rp[3];
	out[tid+8] = rp[2] * rp[2];
}
 /*
 	S T1, T2;
	T1 = in[tid];
	T2 = in[tid+1];
	out[tid] = T1;
	out[tid+1] = T2;
	T1 = in[tid+2];
	T2 = in[tid+3];
	out[tid+2] = T1;
	out[tid+3] = T2;*/
	// if (tid >= poly_size/4) return;
	// S rp[9] = {S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one()}; //TODO: generalize - make template version
	// rp[0] = in[tid + 0];
	// rp[2] = in[tid + 2];
	// rp[1] = in[tid + 1];
	// rp[3] = in[tid + 3];
	// rp[4] = in[tid + 4];
	// rp[5] = in[tid + 5];
	// rp[6] = in[tid + 6];
	// rp[7] = in[tid + 7];
	// rp[8] = in[tid + 8];
	// rp[0] = rp[2];
	// rp[2] = rp[3];
	// rp[1] = rp[4];
	// rp[3] = rp[5];
	// rp[4] = rp[6];
	// rp[5] = rp[7];
	// rp[6] = rp[8];
	// rp[7] = rp[0];
	// rp[8] = rp[1];
	// out[tid] = rp[0];
	// out[tid + 1] = rp[2];
	// out[tid + 2] = rp[1];
	// out[tid + 3] = rp[3];
	// out[tid + 4] = rp[4];
	// out[tid + 5] = rp[5];
	// out[tid + 6] = rp[6];
	// out[tid + 7] = rp[7];
	// out[tid + 8] = rp[8];

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void mult_and_combine_double(S* in, S* out, int poly_size, int poly_shift, int nof_polys, S alpha1, S alpha2){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= poly_size/4) return;
	S rp[9] = {S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one()}; //TODO: generalize - make template version
	S e1, e2, e3, e4, f1, f2, f3, f4;
	#pragma unroll 1
	for (int l = 0; l < nof_polys; l++)
	{
		f1 = in[l*poly_shift + tid];
		f2 = in[l*poly_shift + tid + 2*poly_size];
		f3 = in[l*poly_shift + tid + poly_size];
		f4 = in[l*poly_shift + tid + 3*poly_size];
		e1 = f1 + alpha1 * (f3 - f1) + alpha2 * (f2 - f1) + alpha1 * alpha2 * (f1 + f4 - f2 - f3);
		f1 = in[l*poly_shift + 2*poly_size/4 + tid];
		f2 = in[l*poly_shift + 2*poly_size/4 + tid + 2*poly_size];
		f3 = in[l*poly_shift + 2*poly_size/4 + tid + poly_size];
		f4 = in[l*poly_shift + 2*poly_size/4 + tid + 3*poly_size];
		e2 = f1 + alpha1 * (f3 - f1) + alpha2 * (f2 - f1) + alpha1 * alpha2 * (f1 + f4 - f2 - f3);
		f1 = in[l*poly_shift + poly_size/4 + tid];
		f2 = in[l*poly_shift + poly_size/4 + tid + 2*poly_size];
		f3 = in[l*poly_shift + poly_size/4 + tid + poly_size];
		f4 = in[l*poly_shift + poly_size/4 + tid + 3*poly_size];
		e3 = f1 + alpha1 * (f3 - f1) + alpha2 * (f2 - f1) + alpha1 * alpha2 * (f1 + f4 - f2 - f3);
		f1 = in[l*poly_shift + 3*poly_size/4 + tid];
		f2 = in[l*poly_shift + 3*poly_size/4 + tid + 2*poly_size];
		f3 = in[l*poly_shift + 3*poly_size/4 + tid + poly_size];
		f4 = in[l*poly_shift + 3*poly_size/4 + tid + 3*poly_size];
		e4 = f1 + alpha1 * (f3 - f1) + alpha2 * (f2 - f1) + alpha1 * alpha2 * (f1 + f4 - f2 - f3);
		in[l*poly_shift + tid] = e1;
		in[l*poly_shift + tid + poly_size/2] = e2;
		in[l*poly_shift + tid + poly_size/4] = e3;
		in[l*poly_shift + tid + 3*poly_size/4] = e4;
		rp[0] = l? rp[0]*e1 : e1; //k=0,0
		rp[1] = l? rp[1]*e2 : e2; //k=0,1
		rp[2] = l? rp[2]*e3 : e3; //k=1,0
		rp[3] = l? rp[3]*e4 : e4; //k=1,1
		if (nof_polys == 1) continue;
		rp[4] = l? rp[4]*(e2+e2-e1) : (e2+e2-e1); //k=0,2
		rp[5] = l? rp[5]*(e3+e3-e1) : (e3+e3-e1); //k=2,0
		rp[6] = l? rp[6]*(e4+e4-e3) : (e4+e4-e3); //k=1,2
		rp[7] = l? rp[7]*(e4+e4-e2) : (e4+e4-e2); //k=2,1
		rp[8] = l? rp[8]*(e4+e4+e4+e4+e1-e2-e2-e3-e3) : (e4+e4+e4+e4+e1-e2-e2-e3-e3); //k=2,2
	}
	out[tid] = rp[0];
	out[tid + 1*poly_size/4] = rp[2];
	out[tid + 2*poly_size/4] = rp[1];
	out[tid + 3*poly_size/4] = rp[3];
	if (nof_polys == 1) return;
	out[tid + 4*poly_size/4] = rp[4];
	out[tid + 5*poly_size/4] = rp[5];
	out[tid + 6*poly_size/4] = rp[6];
	out[tid + 7*poly_size/4] = rp[7];
	out[tid + 8*poly_size/4] = rp[8];
}

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void mult_and_combine3(S* in, S* out, int poly_size, int poly_shift, S alpha){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= poly_size/2) return;
	S rp[4] = {S::one(), S::one(), S::one(), S::one()};
	S e1, e2;
	#pragma unroll
	for (int l = 0; l < 3; l++)
	{
		e1 = in[l*poly_shift + tid] + alpha * (in[l*poly_shift + tid + poly_size] - in[l*poly_shift + tid]);
		e2 = in[l*poly_shift + tid + poly_size/2] + alpha * (in[l*poly_shift + tid + poly_size/2 + poly_size] - in[l*poly_shift + tid + poly_size/2]);
		in[l*poly_shift + tid] = e1;
		in[l*poly_shift + tid + poly_size/2] = e2;
		rp[0] = rp[0]*e1;
		rp[1] = rp[1]*e2;
		rp[2] = rp[2]*(e2 + e2 - e1);
		rp[3] = rp[3]*(e2 + e2 + e2 - e1 - e1);
	}
	out[tid] = rp[0];
	out[tid + 1*poly_size/2] = rp[1];
	out[tid + 2*poly_size/2] = rp[2];
	out[tid + 3*poly_size/2] = rp[3];
}

template <typename S>
// __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
__global__ void mult_and_combine(S* in, S* out, int poly_size, int poly_shift, int nof_polys, S alpha){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= poly_size/2) return;
	S rp[5] = {S::one(), S::one(), S::one(), S::one(), S::one()}; //TODO: generalize
	S e1, e2;
	#pragma unroll
	for (int l = 0; l < nof_polys; l++)
	{
		e1 = in[l*poly_shift + tid] + alpha * (in[l*poly_shift + tid + poly_size] - in[l*poly_shift + tid]);
		e2 = in[l*poly_shift + tid + poly_size/2] + alpha * (in[l*poly_shift + tid + poly_size/2 + poly_size] - in[l*poly_shift + tid + poly_size/2]);
		in[l*poly_shift + tid] = e1;
		in[l*poly_shift + tid + poly_size/2] = e2;
		rp[0] = rp[0]*e1;
		rp[1] = rp[1]*e2;
		if (nof_polys == 1) continue;
		rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
		if (nof_polys == 2) continue;
		rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
		if (nof_polys == 3) continue;
		rp[4] = l? rp[4]*(e2 + e2 + e2 + e2 - e1 - e1 - e1) : (e2 + e2 + e2 + e2 - e1 - e1 - e1); //k=4
	}
	out[tid] = rp[0];
	out[tid + 1*poly_size/2] = rp[1];
	if (nof_polys == 1) return;
	out[tid + 2*poly_size/2] = rp[2];
	if (nof_polys == 2) return;
	out[tid + 3*poly_size/2] = rp[3];
	if (nof_polys == 3) return;
	out[tid + 4*poly_size/2] = rp[4];
}

// template <typename S, int M>
// // __global__ void combinations_kernel(S* in, S* out, S (*combine_func)()){
// __global__ void combinations_kernel(S* in, S* out){
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	S rp = s::one;
// 	#pragma unroll
// 	for (int k = 0; k < M+1; k++)
// 	{
// 		#pragma unroll
// 		for (int l = 0; l < M; i++)
// 		{
// 			rp *= in[2*tid] * (1 - k) + in[2*tid + 1] * k;
// 		}
// 	}
// }

// template <typename S>
// __device__ S simple_combine(S* f){
// 	return f[0]*f[1]*f[2]
// }

template <typename S>
S my_hash(){
	S val = S::one() + S::one();
	val = val + val;
	val = val + val; 
	return val + S::one() + S::one();
}

template <typename S>
void sumcheck_alg1(S* evals, S* t, S* T, S C, int n, bool reorder, hipStream_t stream){
	if (reorder) reorder_digits_inplace_and_normalize_kernel<<<1<<(max(n-6,0)),64,0,stream>>>(evals, n, false, mxntt::eRevType::NaturalToRev, false, S::one());
	// S alpha = 1;
	// S alpha = S::one();
	S alpha = my_hash<S>();
	// S alpha = S::rand_host();
  // S alpha = my_hash(/*T, C*/);
  // S rp_even, rp_odd;
  for (int p = 0; p < n-1; p++)
  {
    int nof_threads = 1<<(n-1-p);
		printf("reg nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
    accumulate(evals, t, n-p, 2, 1, stream); //accumulation
		// hipDeviceSynchronize();
		// printf("cuda a err %d\n", hipGetLastError());
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, 2);
		// hipDeviceSynchronize();
		// printf("cuda t err %d\n", hipGetLastError());
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		int NOF_THREADS = min(256,nof_threads);
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, 1<<(n-p), 0, 1); //phase 3
		// hipDeviceSynchronize();
		// printf("cuda err u %d\n", hipGetLastError());
		#ifdef DEBUG
		break;
		#endif
  }
	add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n-1, 2);
}

template <typename S>
void sumcheck_alg1_unified(S* evals, S* t, S* T, S C, int n, bool reorder, hipStream_t stream){
	if (reorder) reorder_digits_inplace_and_normalize_kernel<<<1<<(max(n-6,0)),64,0,stream>>>(evals, n, false, mxntt::eRevType::NaturalToRev, false, S::one());
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	S alpha = my_hash<S>();
	// S alpha = S::rand_host();
  // S alpha = my_hash(/*T, C*/);
  // S rp_even, rp_odd;
  for (int p = 0; p < n-1; p++)
  // for (int p = 0; p < 2; p++)
  {
    int nof_threads = 1<<(n-1-p);
		// printf("nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
    if (p) mult_and_accumulate(evals, t, n-p, alpha, 2, stream); //accumulation
		else accumulate(evals, t, n-p, 2, 1, stream);
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, 2);
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// int NOF_THREADS = 256;
		// int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha); //phase 3
		#ifdef DEBUG
		if (p) break;
		#endif
  }
	#ifndef DEBUG
	update_evals_kernel<<<1, 2,0, stream>>>(evals, alpha, 4, 0, 1);
	#endif
	add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n-1, 2);
}

template <typename S>
void sumcheck_alg3_poly3(S* evals, S* t, S* T, S C, int n, bool reorder, hipStream_t stream){
	if (reorder) reorder_digits_inplace_and_normalize_kernel<<<1<<(max(n-6,0)),64,0,stream>>>(evals, n, false, mxntt::eRevType::NaturalToRev, false, S::one());
	// S alpha = 1;
	// S alpha = S::one();
	S alpha = my_hash<S>();
	// S alpha = S::rand_host();
  // S alpha = my_hash(/*T, C*/);
  // S rp_even, rp_odd;
  for (int p = 0; p < n; p++)
  {
    int nof_threads = 1<<(n-1-p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		// printf("nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
		combinations_kernel3<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n);
		// hipDeviceSynchronize();
		// printf("cuda err u %d\n", hipGetLastError());
		if (p != n-1) accumulate(t, t, n-p, 4, 1, stream);
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, 4);
		// hipDeviceSynchronize();
		// printf("cuda err u %d\n", hipGetLastError());
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		nof_threads = 3<<(n-1-p);
		NOF_THREADS = 64;
		NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    if (p != n-1) update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, 1<<(n-p), 1<<n, 3); //phase 3
		// hipDeviceSynchronize();
		// printf("cuda err u %d\n", hipGetLastError());
		// S h_evals_temp[64*3];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * (64*3), hipMemcpyDeviceToHost);
		// if (p){
		// printf("round %d evals:\n",p);
		// for (int i = 0; i < 64*3; i++)
		// {
		// 	if (i % 64 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
  }
	// update_evals_kernel<<<1, 2,0, stream>>>(evals, alpha);
	// add_to_trace<<<1,1,0,stream>>>(T, evals, 1<<n, n-1, 4);
}

template <typename S>
void sumcheck_alg3_poly3_unified(S* evals, S* t, S* T, S C, int n, hipStream_t stream){
	// S alpha = 1;
	// S alpha = S::one();
	// S alpha = S::rand_host();
  S alpha = my_hash<S>();
  // S rp_even, rp_odd;
  for (int p = 0; p < n; p++)
  {
    int nof_threads = 1<<(n-1-p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		// printf("nof threads %d\n", nof_threads);
    // move update kernel here and unify
    // reduction_kernel<<<nof_threads>>>(evals, t, n-p); //accumulation
		if (p) mult_and_combine3<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n, alpha);
		else combinations_kernel3<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n);
		accumulate(t, t, n-p, 4, 1, stream);
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, 4);
		// hipDeviceSynchronize();
		// printf("cuda err u %d\n", hipGetLastError());
		// S h_evals_temp[8*3];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * (8*3), hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",p);
		// for (int i = 0; i < 8*3; i++)
		// {
		// 	if (i % 8 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, nof_threads); //phase 3
  }
	// update_evals_kernel<<<1, 2,0, stream>>>(evals, alpha, 2);
	// add_to_trace<<<1,1,0,stream>>>(T, evals, n-1, 4);
}


template <typename S>
void sumcheck_generic_unified(S* evals, S* t, S* T, S C, int n, int nof_polys, hipStream_t stream){
	// S alpha = 1;
	// S alpha = S::one();
	// S alpha = S::rand_host();
  S alpha = my_hash<S>();
  // S alpha = S::zero();
  // S rp_even, rp_odd;
  for (int p = 0; p < n; p++)
  {
		// alpha = p%2? S::zero() : S::one();
    int nof_threads = 1<<(n-1-p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		if (nof_polys == 1){
		  if (p) mult_and_accumulate(evals, t, n-p, alpha, 2, stream); //accumulation
			else accumulate(evals, t, n-p, 2, 1, stream);
			// hipDeviceSynchronize();
			// printf("cuda err a %d\n", hipGetLastError());
			if (p == n-1) break;
		}
		else {
			if (p) mult_and_combine<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n, nof_polys, alpha);
			else combinations_kernel<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n, nof_polys);
			// hipDeviceSynchronize();
			// printf("cuda err b %d\n", hipGetLastError());
			accumulate(t, t, n-p, nof_polys+1, 1, stream);
			// hipDeviceSynchronize();
			// printf("cuda err c %d\n", hipGetLastError());
		}
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, nof_polys+1);
		// hipDeviceSynchronize();
		// printf("cuda err d %d\n", hipGetLastError());
		// S h_evals_temp[16*2];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * 16*2, hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",p);
		// for (int i = 0; i < 16*2; i++)
		// {
		// 	if (i % 16 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, nof_threads); //phase 3
  }
	if (nof_polys == 1){
		if (n>1) update_evals_kernel<<<1, 2,0, stream>>>(evals, alpha, 4, 0, 1);
		add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n-1, 2);
	}
}

template <typename S>
void sumcheck_double_round_unified(S* evals, S* t, S* T, S C, int n, int nof_polys, hipStream_t stream){
	// S alpha = 1;
	// S alpha = S::one();
	// S alpha = S::rand_host();
	S alpha1 = my_hash<S>();
  S alpha2 = my_hash<S>() + my_hash<S>();
	// S alpha1 = S::zero();
	// S alpha2 = S::zero();
  // S alpha = S::zero();
  // S rp_even, rp_odd;
  for (int p = 0; p < n/2; p++)
  {
		// alpha = p%2? S::zero() : S::one();
    int nof_threads = 1<<(n-2-2*p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		if (nof_polys == 1){
		  if (p) mult_and_accumulate_double(evals, t, n-2*p, alpha1, alpha2, 2, stream); //accumulation
			else accumulate(evals, t, n-2*p, 2, 2, stream);
			// hipDeviceSynchronize();
			// printf("cuda err a %d\n", hipGetLastError());
			if (p == n/2-1) break;
		}
		else {
			if (p) mult_and_combine_double<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-2*p), 1<<n, nof_polys, alpha1, alpha2);
			else combinations_double_kernel<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-2*p), 1<<n, nof_polys);
			// hipDeviceSynchronize();
			// printf("cuda err b %d\n", hipGetLastError());
			accumulate(t, t, n-2*p, (nof_polys+1)*(nof_polys+1), 2, stream);
			// hipDeviceSynchronize();
			// printf("cuda err c %d\n", hipGetLastError());
		}
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-2-2*p), p, (nof_polys+1)*(nof_polys+1));
		// hipDeviceSynchronize();
		// printf("cuda err d %d\n", hipGetLastError());
		// S h_evals_temp[16*2];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * 16*2, hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",p);
		// for (int i = 0; i < 16*2; i++)
		// {
		// 	if (i % 16 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, nof_threads); //phase 3
  }
	if (nof_polys == 1){
		if (n>2) update_evals_double_kernel<<<1, 4,0, stream>>>(evals, alpha1, alpha2, 16, 0, 1);
		add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n/2-1, 4);
		// S h_evals_temp[16];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * 16, hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",n/2-1);
		// for (int i = 0; i < 16; i++)
		// {
		// 	if (i % 16 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
	}
}

template <typename S>
void sumcheck_double_round_separate(S* evals, S* t, S* T, S C, int n, int nof_polys, hipStream_t stream){
	// S alpha = 1;
	// S alpha = S::one();
	// S alpha = S::rand_host();
	S alpha1 = my_hash<S>();
  S alpha2 = my_hash<S>() + my_hash<S>();
	// S alpha1 = S::zero();
	// S alpha2 = S::zero();
  // S alpha = S::zero();
  // S rp_even, rp_odd;
  for (int p = 0; p < n/2; p++)
  {
		// alpha = p%2? S::zero() : S::one();
    int nof_threads = 1<<(n-2-2*p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		if (nof_polys == 1){
			accumulate(evals, t, n-2*p, 2, 2, stream);
			// hipDeviceSynchronize();
			// printf("cuda err a %d\n", hipGetLastError());
			if (p == n/2-1) break;
		}
		else {
			combinations_double_kernel<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-2*p), 1<<n, nof_polys);
			// combinations_double_test<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t);
			// hipDeviceSynchronize();
			// printf("cuda err b %d\n", hipGetLastError());
			accumulate(t, t, n-2*p, (nof_polys+1)*(nof_polys+1), 2, stream);
			// hipDeviceSynchronize();
			// printf("cuda err c %d\n", hipGetLastError());
		}
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-2-2*p), p, (nof_polys+1)*(nof_polys+1));

		nof_threads = nof_polys<<(n-2-2*p);
		NOF_THREADS = 64;
		NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    if (p != n/2-1) update_evals_double_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha1, alpha2, 1<<(n-2*p), 1<<n, nof_polys); //phase 3
		// hipDeviceSynchronize();
		// printf("cuda err d %d\n", hipGetLastError());
		// S h_evals_temp[16*2];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * 16*2, hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",p);
		// for (int i = 0; i < 16*2; i++)
		// {
		// 	if (i % 16 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, nof_threads); //phase 3
  }
	if (nof_polys == 1){
		// update_evals_double_kernel<<<1, 4,0, stream>>>(evals, alpha1, alpha2, 16, 0, 1);
		add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n/2-1, 4);
		// S h_evals_temp[16];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * 16, hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",n/2-1);
		// for (int i = 0; i < 16; i++)
		// {
		// 	if (i % 16 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
	}
}

template <typename S>
void sumcheck_generic_separate(S* evals, S* t, S* T, S C, int n, int nof_polys, hipStream_t stream){
	// S alpha = 1;
	// S alpha = S::one();
	// S alpha = S::rand_host();
	S alpha = my_hash<S>();
	// S alpha1 = S::zero();
	// S alpha2 = S::zero();
  // S alpha = S::zero();
  // S rp_even, rp_odd;
  for (int p = 0; p < n; p++)
  {
		// alpha = p%2? S::zero() : S::one();
    int nof_threads = 1<<(n-1-p);
		int NOF_THREADS = 64;
		int NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
		if (nof_polys == 1){
			accumulate(evals, t, n-p, 2, 1, stream);
			// hipDeviceSynchronize();
			// printf("cuda err a %d\n", hipGetLastError());
			if (p == n-1) break;
		}
		else {
			combinations_kernel<<<NOF_BLOCKS, NOF_THREADS,0,stream>>>(evals, t, 1<<(n-p), 1<<n, nof_polys);
			// hipDeviceSynchronize();
			// printf("cuda err b %d\n", hipGetLastError());
			accumulate(t, t, n-p, nof_polys+1, 1, stream);
			// hipDeviceSynchronize();
			// printf("cuda err c %d\n", hipGetLastError());
		}
		add_to_trace<<<1,1,0,stream>>>(T, t, 1<<(n-1-p), p, nof_polys+1);

		nof_threads = nof_polys<<(n-1-p);
		NOF_THREADS = 64;
		NOF_BLOCKS = (nof_threads + NOF_THREADS - 1) / NOF_THREADS;
    if (p != n-1) update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, 1<<(n-p), 1<<n, nof_polys); //phase 3
		// hipDeviceSynchronize();
		// printf("cuda err d %d\n", hipGetLastError());
		// S h_evals_temp[16*2];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * 16*2, hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",p);
		// for (int i = 0; i < 16*2; i++)
		// {
		// 	if (i % 16 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
    // T[2*p+1] = t[0];
    // T[2*p+2] = t[1];
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
    // update_evals_kernel<<<NOF_BLOCKS, NOF_THREADS,0, stream>>>(evals, alpha, nof_threads); //phase 3
  }
	if (nof_polys == 1){
		// update_evals_double_kernel<<<1, 4,0, stream>>>(evals, alpha1, alpha2, 16, 0, 1);
		add_to_trace<<<1,1,0,stream>>>(T, evals, 1, n-1, 2);
		// S h_evals_temp[16];
		// hipMemcpy(h_evals_temp, evals, sizeof(S) * 16, hipMemcpyDeviceToHost);
		// if (1){
		// printf("round %d evals:\n",n/2-1);
		// for (int i = 0; i < 16; i++)
		// {
		// 	if (i % 16 == 0) printf("\n");
		// 	std::cout << i << " " << h_evals_temp[i] << std::endl;
		// }
		// }
	}
}

template <typename S>
void sumcheck_alg1_ref(S* evals, S* t, S* T, S C, int n){
  // S alpha = my_hash(/*T, C*/);
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	S alpha = my_hash<S>();
  S rp_bottom, rp_top;
  for (int p = 0; p < n; p++)
  {
		// rp_even = 0; rp_odd = 0;
		rp_bottom = S::zero(); rp_top = S::zero();
		// printf("evals\n");
		// for (int i = 0; i < 1<<(n-p); i++)
		// {
		// 	printf("%d, ",evals[i]);
		// }
		// printf("\n");
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			rp_bottom = rp_bottom + evals[i];
			rp_top = rp_top + evals[i+(1<<(n-1-p))];
		}
    T[2*p+1] = rp_bottom;
    T[2*p+2] = rp_top;
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// alpha = 1;
		// alpha = S::one();
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			t[i] = (S::one() - alpha) * evals[i] + alpha * evals[i+(1<<(n-1-p))];
			// t[i] = (1-alpha)*evals[2*i] + alpha*evals[2*i+1];
		}
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			evals[i] = t[i];
		}
  }
}

template <typename S>
void sumcheck_alg3_ref(S* evals, S* t, S* T, S C, int n){
  // S alpha = my_hash(/*T, C*/);
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	S alpha = my_hash<S>();
  
  for (int p = 0; p < n; p++)
  {

		// rp_even = 0; rp_odd = 0;
		// printf("evals\n");
		// for (int i = 0; i < 1<<(n-p); i++)
		// {
		// 	printf("%d, ",evals[i]);
		// }
		// printf("\n");
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			S rp[4] = {S::one(), S::one(), S::one(), S::one()};
			for (int l = 0; l < 3; l++)
			{
				S e1 = evals[(l<<(n-p)) + i];
				S e2 = evals[(l<<(n-p)) + i + (1<<(n-1-p))];
				rp[0] = l? rp[0]*e1 : e1; //k=0
				rp[1] = l? rp[1]*e2 : e2; //k=1
				rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
				// rp[3] = l? rp[3]*(e1 + e1 - e2) : (e1 + e1 - e2); //k=-1
				rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
			}
			T[4*p+1] = T[4*p+1] + rp[0];
			T[4*p+2] = T[4*p+2] + rp[1];
			T[4*p+3] = T[4*p+3] + rp[2];
			T[4*p+4] = T[4*p+4] + rp[3];
		}
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// alpha = 1;
		// alpha = S::one();
		for (int l = 0; l < 3; l++)
		{
			for (int i = 0; i < 1<<(n-1-p); i++)
			{
				t[(l<<(n-1-p)) + i] = (S::one() - alpha) * evals[(l<<(n-p)) + i] + alpha * evals[(l<<(n-p)) + i + (1<<(n-1-p))];
				// t[i] = (1-alpha)*evals[2*i] + alpha*evals[2*i+1];
			}
		}
// 		if (1)
// {		printf("ref round %d evals:\n",p);
// 		for (int i = 0; i < 3<<(n-p); i++)
// 		{
// 			std::cout << i << " " << evals[i] << std::endl;
// 		}}
		for (int i = 0; i < 3<<(n-1-p); i++)
		{
			evals[i] = t[i];
		}
  }
}

template <typename S>
void sumcheck_generic_ref(S* evals, S* t, S* T, S C, int n, int nof_polys){
  // S alpha = my_hash(/*T, C*/);
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	S alpha = my_hash<S>();
  
  for (int p = 0; p < n; p++)
  {

		// rp_even = 0; rp_odd = 0;
		// printf("evals\n");
		// for (int i = 0; i < 1<<(n-p); i++)
		// {
		// 	printf("%d, ",evals[i]);
		// }
		// printf("\n");
		for (int i = 0; i < 1<<(n-1-p); i++)
		{
			S rp[5] = {S::one(), S::one(), S::one(), S::one(), S::one()};
			for (int l = 0; l < nof_polys; l++)
			{
				S e1 = evals[(l<<(n-p)) + i];
				S e2 = evals[(l<<(n-p)) + i + (1<<(n-1-p))];
				rp[0] = l? rp[0]*e1 : e1; //k=0
				rp[1] = l? rp[1]*e2 : e2; //k=1
				if (nof_polys > 1) rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
				if (nof_polys > 2) rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
				if (nof_polys > 3) rp[4] = l? rp[4]*(e2 + e2 + e2 + e2 - e1 - e1 - e1) : (e2 + e2 + e2 + e2 - e1 - e1 - e1); //k=4
			}
			T[(nof_polys+1)*p+1] = T[(nof_polys+1)*p+1] + rp[0];
			T[(nof_polys+1)*p+2] = T[(nof_polys+1)*p+2] + rp[1];
			if (nof_polys > 1) T[(nof_polys+1)*p+3] = T[(nof_polys+1)*p+3] + rp[2];
			if (nof_polys > 2) T[(nof_polys+1)*p+4] = T[(nof_polys+1)*p+4] + rp[3];
			if (nof_polys > 3) T[(nof_polys+1)*p+5] = T[(nof_polys+1)*p+5] + rp[4];
		}
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// alpha = 1;
		// alpha = S::one();
		for (int l = 0; l < nof_polys; l++)
		{
			for (int i = 0; i < 1<<(n-1-p); i++)
			{
				t[(l<<(n-1-p)) + i] = (S::one() - alpha) * evals[(l<<(n-p)) + i] + alpha * evals[(l<<(n-p)) + i + (1<<(n-1-p))];
				// t[i] = (1-alpha)*evals[2*i] + alpha*evals[2*i+1];
			}
		}
// 		if (1)
// {		printf("ref round %d evals:\n",p);
// 		for (int i = 0; i < 3<<(n-p); i++)
// 		{
// 			std::cout << i << " " << evals[i] << std::endl;
// 		}}
		for (int i = 0; i < nof_polys<<(n-1-p); i++)
		{
			evals[i] = t[i];
		}
  }
}

template <typename S>
void sumcheck_double_round_ref(S* evals, S* t, S* T, S C, int n, int nof_polys){
  // S alpha = my_hash(/*T, C*/);
	// S alpha = 1;
	// S alpha = S::one() + S::one();
	// S alpha = my_hash<S>();
	S alpha1 = my_hash<S>();
  S alpha2 = my_hash<S>() + my_hash<S>();
	// S alpha1 = S::zero();
	// S alpha2 = S::zero();
	// S alpha2 = S::zero();
  
  for (int p = 0; p < n/2; p++)
  {

		// rp_even = 0; rp_odd = 0;
		// printf("evals\n");
		// for (int i = 0; i < 1<<(n-p); i++)
		// {
		// 	printf("%d, ",evals[i]);
		// }
		// printf("\n");
		for (int i = 0; i < 1<<(n-2-2*p); i++)
		{
			S rp[9] = {S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one(), S::one()};
			for (int l = 0; l < nof_polys; l++)
			{
				S e1 = evals[(l<<(n-2*p)) + i];
				S e2 = evals[(l<<(n-2*p)) + i + (1<<(n-1-2*p))];
				S e3 = evals[(l<<(n-2*p)) + i + (1<<(n-2-2*p))];
				S e4 = evals[(l<<(n-2*p)) + i + (1<<(n-1-2*p)) + (1<<(n-2-2*p))];
				//e1 + k1 * (e3 - e1) + k2 * (e2 - e1) + k1 * k2 * (e1 + e4 - e2 - e3);
				rp[0] = l? rp[0]*e1 : e1; //k=0,0
				rp[1] = l? rp[1]*e2 : e2; //k=0,1
				rp[2] = l? rp[2]*e3 : e3; //k=1,0
				rp[3] = l? rp[3]*e4 : e4; //k=1,1
				if (nof_polys == 1) continue;
				rp[4] = l? rp[4]*(e2+e2-e1) : (e2+e2-e1); //k=0,2
				rp[5] = l? rp[5]*(e3+e3-e1) : (e3+e3-e1); //k=2,0
				rp[6] = l? rp[6]*(e4+e4-e3) : (e4+e4-e3); //k=1,2
				rp[7] = l? rp[7]*(e4+e4-e2) : (e4+e4-e2); //k=2,1
				rp[8] = l? rp[8]*(e4+e4+e4+e4+e1-e2-e2-e3-e3) : (e4+e4+e4+e4+e1-e2-e2-e3-e3); //k=2,2
				// if (nof_polys > 1) rp[2] = l? rp[2]*(e2 + e2 - e1) : (e2 + e2 - e1); //k=2
				// if (nof_polys > 2) rp[3] = l? rp[3]*(e2 + e2 + e2 - e1 - e1) : (e2 + e2 + e2 - e1 - e1); //k=3
				// if (nof_polys > 3) rp[4] = l? rp[4]*(e2 + e2 + e2 + e2 - e1 - e1 - e1) : (e2 + e2 + e2 + e2 - e1 - e1 - e1); //k=4
			}
			T[(nof_polys+1)*(nof_polys+1)*p+1] = T[(nof_polys+1)*(nof_polys+1)*p+1] + rp[0];
			T[(nof_polys+1)*(nof_polys+1)*p+2] = T[(nof_polys+1)*(nof_polys+1)*p+2] + rp[2];
			T[(nof_polys+1)*(nof_polys+1)*p+3] = T[(nof_polys+1)*(nof_polys+1)*p+3] + rp[1];
			T[(nof_polys+1)*(nof_polys+1)*p+4] = T[(nof_polys+1)*(nof_polys+1)*p+4] + rp[3];
			if (nof_polys > 1) {
				T[(nof_polys+1)*(nof_polys+1)*p+5] = T[(nof_polys+1)*(nof_polys+1)*p+5] + rp[4];
				T[(nof_polys+1)*(nof_polys+1)*p+6] = T[(nof_polys+1)*(nof_polys+1)*p+6] + rp[5];
				T[(nof_polys+1)*(nof_polys+1)*p+7] = T[(nof_polys+1)*(nof_polys+1)*p+7] + rp[6];
				T[(nof_polys+1)*(nof_polys+1)*p+8] = T[(nof_polys+1)*(nof_polys+1)*p+8] + rp[7];
				T[(nof_polys+1)*(nof_polys+1)*p+9] = T[(nof_polys+1)*(nof_polys+1)*p+9] + rp[8];
			}
			// if (nof_polys > 1) T[(nof_polys+1)*p+3] = T[(nof_polys+1)*p+3] + rp[2];
			// if (nof_polys > 2) T[(nof_polys+1)*p+4] = T[(nof_polys+1)*p+4] + rp[3];
			// if (nof_polys > 3) T[(nof_polys+1)*p+5] = T[(nof_polys+1)*p+5] + rp[4];
		}
    // alpha = my_hash(/*alpha, t[0], t[1]*/); //phase 2
		// alpha = 1;
		// alpha = S::one();
		for (int l = 0; l < nof_polys; l++)
		{
			for (int i = 0; i < 1<<(n-2-2*p); i++)
			{
				S e1 = evals[(l<<(n-2*p)) + i];
				S e2 = evals[(l<<(n-2*p)) + i + (1<<(n-1-2*p))];
				S e3 = evals[(l<<(n-2*p)) + i + (1<<(n-2-2*p))];
				S e4 = evals[(l<<(n-2*p)) + i + (1<<(n-1-2*p)) + (1<<(n-2-2*p))];
				t[(l<<(n-2-2*p)) + i] = e1 + alpha1 * (e3 - e1) + alpha2 * (e2 - e1) + alpha1 * alpha2 * (e1 + e4 - e2 - e3);
				// t[i] = (1-alpha)*evals[2*i] + alpha*evals[2*i+1];
			}
		}
// 		if (1)
// {		printf("ref round %d evals:\n",p);
// 		for (int i = 0; i < nof_polys<<(n-2*p); i++)
// 		{
// 			std::cout << i << " " << evals[i] << std::endl;
// 		}}
		for (int i = 0; i < nof_polys<<(n-2-2*p); i++)
		{
			evals[i] = t[i];
		}
  }
}